
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

// Great help:
// http://www.pages.drexel.edu/~cfa22/msim/node11.html
// gcc -O3 -o name name.c


__global__ void Ising(int* G, int* L,int n){
      int i = threadIdx.x/n;
      int j = threadIdx.x%n;
      int x = G[(i-1+n)%n*n+j] + G[i*n+(j-1+n)%n] + G[i*n+j] + G[(i+1)%n*n+j] + G[i*n+(j+1)%n];
      L[i*n+j] = (x>0) - (x<0);
      printf(" I am %d, x is %d, L is %d\n", threadIdx.x, x,   L[i*n+j]);
}


void init ( int* F, int L) {
  int i,j;
  for (i=0;i<L;i++) {
    for (j=0;j<L;j++) {
      F[i*L+j]=2*(rand()%2) - 1;
    }
  }
}

void printThatShit(int* G,int n){
    for(int i=0; i < n ; i++){
        for(int j=0; j < n ; j++){
            printf("%d ",G[i*n+j]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[]){

    int n ,k;
    srand((unsigned int)time(NULL));

    // pairnw diastash kai iteration
    if(argc < 3){
		printf("bale 2 orismata re lulu, arithmo iteration kai diastash");
		return 0;
	}

    k = (int) strtol(argv[1],NULL,10);
    n = (int) strtol(argv[2],NULL,10);

    // pairnw kati pipes gia to pws bgazei tuxaia 0,1 kai to kanei -1 kai 1
    int* F = (int*)malloc(n * n * sizeof(int));
    int* L = (int*)malloc(n * n * sizeof(int));

    int *d_F, *d_L;
    hipMalloc(&d_F, n * n * sizeof(int));
    hipMalloc(&d_L, n * n * sizeof(int));

    // balw tis tuxaies times ston F
    init(F,n);

    hipMemcpy(d_F, F, n * n * sizeof(int), hipMemcpyHostToDevice);

    // kaloume edw thn fash
    for(int i = 0; i < k ; i++ ){
        Ising<<<1,n*n>>>(d_F,d_L,n);
        int* temp = d_F;
        d_F = d_L;
        d_L = temp;
    }

    hipMemcpy(F, d_F, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // Print That Shit
    printThatShit(F,n);
    printThatShit(L,n);


    // Free everything
    free(L);
    free(F);
    hipFree(d_F);
    hipFree(d_L);

}
